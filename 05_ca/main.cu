#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <assert.h>
#include "Event.h"
#include "GPUHitsAndDoublets.h"
#include "GPUCACell.h"
#include "parser.h"
#include "hip/hip_runtime.h"
#include "kernels.h"

static void show_usage(std::string name)
{
    std::cerr << "\nUsage: " << name << " <option(s)>" << " Options:\n"
            << "\t-h,--help\t\tShow this help message\n"
            << "\t-n <number of events>\tSpecify the number of events to process [default: as many there are in the file]\n"
            << "\t-i <inputFile>\tSpecify the path of the input ASCII file containing the events to process [default: ../input/parsed.out]\n"
            << std::endl;

}

int main(int argc, char** argv)
{

    if (argc < 2)
    {
        show_usage(argv[0]);
        return 1;
    }

    unsigned int maxEvents = 10;
    std::string inputFile = "../input/parsed_noPU_fix.txt";
    unsigned int numberOfCUDAStreams = 5;
    unsigned int numberOfEventsPerStreamPerIteration = 1;

    for (int i = 1; i < argc; ++i)
    {
        std::string arg = argv[i];
        if ((arg == "-h") || (arg == "--help"))
        {
            show_usage(argv[0]);
            return 0;
        }
        else if (arg == "-n")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> maxEvents))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }
        else if (arg == "-i")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> inputFile))
                {
                    std::cerr << "Invalid string " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

        else if (arg == "-s")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> numberOfCUDAStreams))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

        else if (arg == "-b")
        {
            if (i + 1 < argc) // Make sure we aren't at the end of argv!
            {
                i++;
                std::istringstream ss(argv[i]);
                if (!(ss >> numberOfEventsPerStreamPerIteration))
                {
                    std::cerr << "Invalid number " << argv[i] << '\n';
                    exit(1);

                }
            }
        }

    }

    std::vector<HostEvent> hostEvents;
    Region* h_regionParams;
    hipHostMalloc(&h_regionParams, sizeof(Region));

    if (maxEvents > 0)
    {
        hostEvents.reserve(maxEvents);
    }

    constexpr unsigned int maxNumberOfQuadruplets = 3000;
    constexpr unsigned int maxCellsPerHit = 100;
    unsigned int maxNumberOfHits = 0;
    unsigned int maxNumberOfDoublets = 0;
    parseinputFile(inputFile, hostEvents, *h_regionParams, maxEvents, maxNumberOfHits,
            maxNumberOfDoublets);
    unsigned int nEvents = hostEvents.size();
    std::cout << "Correctly parsed file containing " << nEvents << " events." << std::endl;

//    std::cout << "max number of hits and doublets in file " << maxNumberOfHits << " "
//            << maxNumberOfDoublets << std::endl;
//
//    std::cout << "hostEvents contains " << hostEvents.size() << " events" <<  std::endl;
//
//    for(auto& ev: hostEvents)
//    {
//        std::cout << "event id " <<ev.eventId << " rootlayers " << ev.rootLayers.size() << " hitsLayers " << ev.hitsLayers.size() <<
//                " layerPairs " <<  ev.doublets.size() << std::endl;
//        for(auto& rl : ev.rootLayers)
//            std::cout<< "root layer: " << rl << std::endl;
//        for(auto& hl : ev.hitsLayers)
//        {
//            std::cout<< "hits on layer: " << hl.size << " " << hl.x.size() << std::endl;
//
//            for(int jj = 0; jj < hl.x.size(); ++jj)
//                std::cout<< "hit " << jj << hl.x[jj] << " " << hl.y[jj] << " " << hl.z[jj] << std::endl;
//
//
//        }
//        for(auto& dl : ev.doublets)
//            {
//                std::cout<< "doublets on layer pair: " << dl.innerLayerId << " " << dl.outerLayerId << " " << dl.size << " " << dl.indices.size() << std::endl;
//                    for(unsigned int i = 0; i<dl.size; ++i )
//                    {
//                        std::cout<< "\t hits in doublet: "<< i << " " <<  dl.indices[2*i] << " "<< dl.indices[2*i+1] <<" " << ev.hitsLayers[dl.innerLayerId].x[dl.indices[2*i]] << " "<< ev.hitsLayers[dl.innerLayerId].y[dl.indices[2*i]] << " "<<ev.hitsLayers[dl.innerLayerId].z[dl.indices[2*i]]<< " "
//                         << " \t\t" << ev.hitsLayers[dl.outerLayerId].x[dl.indices[2*i+1]] << " "<< ev.hitsLayers[dl.outerLayerId].y[dl.indices[2*i+1]] << " "<<ev.hitsLayers[dl.outerLayerId].z[dl.indices[2*i+1]]<< std::endl;
//                    }
//            }
//    }





    std::cout << "preallocating pinned memory on host" << std::endl;

    unsigned int eventsPreallocatedOnGPU = numberOfEventsPerStreamPerIteration
            * numberOfCUDAStreams;
    unsigned int maxNumberOfLayers = 10;
    unsigned int maxNumberOfLayerPairs = 13;
    unsigned int maxNumberOfRootLayerPairs = 3;


    constexpr const float theThetaCut = 0.002f;
    constexpr const float thePhiCut = 0.2f;
    constexpr const float theHardPtCut = 0.0f;


    // HOST ALLOCATIONS FOR THE INPUT
    //////////////////////////////////////
    GPUEvent *h_allEvents;
    unsigned int* h_indices;
    GPULayerDoublets* h_doublets;
    hipHostMalloc(&h_allEvents, nEvents * sizeof(GPUEvent));
    //per each event per each layerPair you have a max number of doublets
    hipHostMalloc(&h_indices, nEvents *maxNumberOfLayerPairs*maxNumberOfDoublets*2* sizeof(int));
    hipHostMalloc(&h_doublets, nEvents *maxNumberOfLayerPairs * sizeof(GPULayerDoublets));

    //per each event per each layer you have a max number of hits x y z
    GPULayerHits* h_layers;
    float *h_x, *h_y, *h_z;
    unsigned int* h_rootLayerPairs;

    hipHostMalloc(&h_layers, nEvents * maxNumberOfLayers * sizeof(GPULayerHits));
    hipHostMalloc(&h_x, nEvents * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipHostMalloc(&h_y, nEvents * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipHostMalloc(&h_z, nEvents * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipHostMalloc(&h_rootLayerPairs, nEvents * maxNumberOfRootLayerPairs * sizeof(int));


    //GPU ALLOCATIONS
    std::cout << "preallocating memory on GPU " << std::endl;
    Region* d_regionParams;

    GPUEvent *d_events;
    unsigned int* d_indices;
    GPULayerDoublets* d_doublets;
    GPULayerHits* d_layers;
    float *d_x, *d_y, *d_z;
    unsigned int* d_rootLayerPairs;

    hipMalloc(&d_regionParams, sizeof(Region));
    hipMemcpy(d_regionParams, h_regionParams, sizeof(Region), hipMemcpyHostToDevice);
    hipMalloc(&d_events, eventsPreallocatedOnGPU * sizeof(GPUEvent));
    hipMalloc(&d_indices, eventsPreallocatedOnGPU *maxNumberOfLayerPairs*maxNumberOfDoublets*2* sizeof(int));
    hipMalloc(&d_doublets, eventsPreallocatedOnGPU *maxNumberOfLayerPairs * sizeof(GPULayerDoublets));
    hipMalloc(&d_layers, eventsPreallocatedOnGPU * maxNumberOfLayers * sizeof(GPULayerHits));
    hipMalloc(&d_x, eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipMalloc(&d_y, eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipMalloc(&d_z, eventsPreallocatedOnGPU * maxNumberOfLayers * maxNumberOfHits * sizeof(float));
    hipMalloc(&d_rootLayerPairs, eventsPreallocatedOnGPU * maxNumberOfRootLayerPairs * sizeof(unsigned int));

    for (unsigned int i = 0; i < nEvents; ++i)
    {
        h_allEvents[i].eventId = hostEvents[i].eventId;

        h_allEvents[i].numberOfRootLayerPairs = 0;

        h_allEvents[i].numberOfLayers = hostEvents[i].hitsLayers.size();
        h_allEvents[i].numberOfLayerPairs = hostEvents[i].doublets.size();
        for (unsigned int j = 0; j < maxNumberOfLayerPairs; ++j)
        {
            auto doubletIdx = i*maxNumberOfLayerPairs+j;
            h_doublets[doubletIdx].size = 0;
        }

        for (unsigned int j = 0; j < maxNumberOfLayers; ++j)
        {
            auto layerIdx = i*maxNumberOfLayers+j;

            h_layers[layerIdx].size = 0;

        }


        for (unsigned int j = 0; j < hostEvents[i].doublets.size(); ++j)
        {
            auto layerPairIndex = i*maxNumberOfLayerPairs+j;

            h_doublets[layerPairIndex].size = hostEvents[i].doublets[j].size;
            h_doublets[layerPairIndex].innerLayerId = hostEvents[i].doublets[j].innerLayerId;
            h_doublets[layerPairIndex].outerLayerId = hostEvents[i].doublets[j].outerLayerId;

            for (unsigned int l = 0; l < hostEvents[i].rootLayers.size(); ++l)
            {
                if(hostEvents[i].rootLayers[l]==h_doublets[layerPairIndex].innerLayerId)
                {
                    auto rootlayerPairId = i*maxNumberOfRootLayerPairs+ h_allEvents[i].numberOfRootLayerPairs;
                    h_rootLayerPairs[rootlayerPairId] = j;
                    h_allEvents[i].numberOfRootLayerPairs++;
                }


            }
            for(unsigned int l = 0; l < hostEvents[i].doublets[j].size; ++l)
            {
                auto hitId = layerPairIndex*maxNumberOfDoublets*2 + 2*l;
                h_indices[hitId] = hostEvents[i].doublets[j].indices[2*l];
                h_indices[hitId+1] = hostEvents[i].doublets[j].indices[2*l+1];

            }

        }





        for (unsigned int j = 0; j < hostEvents[i].hitsLayers.size(); ++j)
        {
            auto layerIdx = i*maxNumberOfLayers+j;

            h_layers[layerIdx].size = hostEvents[i].hitsLayers[j].size;
            h_layers[layerIdx].layerId = hostEvents[i].hitsLayers[j].layerId;
            for(unsigned int l = 0; l < hostEvents[i].hitsLayers[j].size; ++l)
            {
                auto hitId = layerIdx*maxNumberOfHits + l;

                h_x[hitId] = hostEvents[i].hitsLayers[j].x[l];
                h_y[hitId] = hostEvents[i].hitsLayers[j].y[l];
                h_z[hitId] = hostEvents[i].hitsLayers[j].z[l];

            }

        }


    }



    for (unsigned int i = 0; i < nEvents; ++i)
    {
        assert(h_allEvents[i].eventId == hostEvents[i].eventId);
        assert(h_allEvents[i].numberOfLayers == hostEvents[i].hitsLayers.size());
        assert(h_allEvents[i].numberOfLayerPairs == hostEvents[i].doublets.size());
        auto cellId=0;
        for (unsigned int j = 0; j < hostEvents[i].doublets.size();++j)
        {
            auto layerPairIdx = i*maxNumberOfLayerPairs+j;
            assert(h_doublets[layerPairIdx].size == hostEvents[i].doublets[j].size);


            for(unsigned int l = 0; l < hostEvents[i].doublets[j].size; ++l)
            {
                auto hitId = layerPairIdx*maxNumberOfDoublets*2 + 2*l;
                assert(hostEvents[i].doublets[j].indices[2*l] == h_indices[hitId]);
                assert(hostEvents[i].doublets[j].indices[2*l+1] == h_indices[hitId+1]);
                auto innerHitId = hostEvents[i].doublets[j].indices[2*l];
                auto outerHitId = hostEvents[i].doublets[j].indices[2*l+1];
                auto innerLayerId = hostEvents[i].doublets[j].innerLayerId;
                auto outerLayerId = hostEvents[i].doublets[j].outerLayerId;

                auto xinnerIndex = i*maxNumberOfLayers*maxNumberOfHits + maxNumberOfHits*innerLayerId + innerHitId;
                assert(hostEvents[i].hitsLayers[innerLayerId].x[innerHitId]==h_x[xinnerIndex] );
//                float x1, y1, z1, x2, y2, z2;
//
//                x1 = hostEvents[i].hitsLayers[innerLayerId].x[innerHitId];
//                y1 = hostEvents[i].hitsLayers[innerLayerId].y[innerHitId];
//                z1 = hostEvents[i].hitsLayers[innerLayerId].z[innerHitId];
//                x2 = hostEvents[i].hitsLayers[outerLayerId].x[outerHitId];
//                y2 = hostEvents[i].hitsLayers[outerLayerId].y[outerHitId];
//                z2 = hostEvents[i].hitsLayers[outerLayerId].z[outerHitId];
//                printf("\n\n\nCPU cellid %d innerhit outerhit (xyz) (%f %f %f), (%f %f %f)\n",cellId, x1,y1,z1,x2,y2,z2);

                cellId++;
            }

        }

    }

    //////////////////////////////////////////////////////////
    // ALLOCATIONS FOR THE INTERMEDIATE RESULTS (STAYS ON GPU)
    //////////////////////////////////////////////////////////



    GPUCACell *device_theCells;


    hipMalloc(&device_theCells, eventsPreallocatedOnGPU *maxNumberOfLayerPairs * maxNumberOfDoublets*sizeof(GPUCACell ));
    //////////////////////////////////////////////////////////
    // ALLOCATIONS FOR THE RESULTS (STAYS ON GPU)
    //////////////////////////////////////////////////////////


    GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> * d_foundNtuplets;
    GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> * h_foundNtuplets;
    hipMalloc(&d_foundNtuplets, eventsPreallocatedOnGPU*sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ));
    hipHostMalloc(&h_foundNtuplets, eventsPreallocatedOnGPU*sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ));

    GPUSimpleVector<maxCellsPerHit, unsigned int> * device_isOuterHitOfCell;


    hipMalloc(&device_isOuterHitOfCell, eventsPreallocatedOnGPU* maxNumberOfLayers * maxNumberOfHits* sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int>));

    hipMemset(device_isOuterHitOfCell, 0, eventsPreallocatedOnGPU* maxNumberOfLayers * maxNumberOfHits* sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int>));


    hipStream_t streams[numberOfCUDAStreams];
    for(int i = 0; i < numberOfCUDAStreams; ++i)
    {
        hipStreamCreate(&streams[i]);

    }

    //INITIALIZATION IS NOW OVER
    //HERE STARTS THE COMPUTATION

    for (unsigned int i = 0; i < nEvents; ++i)
    {
//        unsigned int i =1;
        unsigned int streamIndex = i%numberOfCUDAStreams;
//        unsigned int streamIndex =1;
        auto d_firstLayerPairInEvt = maxNumberOfLayerPairs*streamIndex;
        auto d_firstLayerInEvt = maxNumberOfLayers*streamIndex;
        auto d_firstDoubletInEvent = d_firstLayerPairInEvt*maxNumberOfDoublets;
        auto d_firstHitInEvent = d_firstLayerInEvt*maxNumberOfHits;

        auto h_firstLayerPairInEvt = maxNumberOfLayerPairs*i;
        auto h_firstLayerInEvt = maxNumberOfLayers*i;
        auto h_firstDoubletInEvent = h_firstLayerPairInEvt*maxNumberOfDoublets;
        auto h_firstHitInEvent = h_firstLayerInEvt*maxNumberOfHits;


        for(unsigned int j= 0; j< h_allEvents[i].numberOfLayerPairs;++j)
        {
            h_doublets[h_firstLayerPairInEvt+j].indices = &d_indices[d_firstDoubletInEvent*2 + j*maxNumberOfDoublets*2];
            hipMemcpyAsync(&d_indices[d_firstDoubletInEvent*2 + j*maxNumberOfDoublets*2],
                    &h_indices[h_firstDoubletInEvent*2+j*maxNumberOfDoublets*2],
                    h_doublets[h_firstLayerPairInEvt+j].size*2*sizeof(int),
                    hipMemcpyHostToDevice, streams[streamIndex]);
        }

        for(unsigned int j= 0; j< h_allEvents[i].numberOfLayers;++j)
        {
            h_layers[h_firstLayerInEvt+j].x = &d_x[d_firstHitInEvent + maxNumberOfHits*j];
            h_layers[h_firstLayerInEvt+j].y = &d_y[d_firstHitInEvent + maxNumberOfHits*j];
            h_layers[h_firstLayerInEvt+j].z = &d_z[d_firstHitInEvent + maxNumberOfHits*j];
            hipMemcpyAsync(h_layers[h_firstLayerInEvt+j].x, &h_x[h_firstHitInEvent+j*maxNumberOfHits], h_layers[h_firstLayerInEvt+j].size*sizeof(float), hipMemcpyHostToDevice, streams[streamIndex]);
            hipMemcpyAsync(h_layers[h_firstLayerInEvt+j].y, &h_y[h_firstHitInEvent+j*maxNumberOfHits], h_layers[h_firstLayerInEvt+j].size*sizeof(float), hipMemcpyHostToDevice, streams[streamIndex]);
            hipMemcpyAsync(h_layers[h_firstLayerInEvt+j].z, &h_z[h_firstHitInEvent+j*maxNumberOfHits], h_layers[h_firstLayerInEvt+j].size*sizeof(float), hipMemcpyHostToDevice, streams[streamIndex]);
        }

        hipMemcpyAsync(&d_rootLayerPairs[maxNumberOfRootLayerPairs*streamIndex], &h_rootLayerPairs[maxNumberOfRootLayerPairs*i], h_allEvents[i].numberOfRootLayerPairs*sizeof(unsigned int), hipMemcpyHostToDevice, streams[streamIndex]);
        hipMemcpyAsync(&d_doublets[d_firstLayerPairInEvt], &h_doublets[h_firstLayerPairInEvt], h_allEvents[i].numberOfLayerPairs*sizeof(GPULayerDoublets), hipMemcpyHostToDevice, streams[streamIndex]);
        hipMemcpyAsync(&d_layers[d_firstLayerInEvt], &h_layers[h_firstLayerInEvt], h_allEvents[i].numberOfLayers*sizeof(GPULayerHits), hipMemcpyHostToDevice, streams[streamIndex]);


        hipMemcpyAsync(&d_events[streamIndex], &h_allEvents[i], sizeof(GPUEvent), hipMemcpyHostToDevice, streams[streamIndex]);

        dim3 numberOfBlocks_create(32, h_allEvents[i].numberOfLayerPairs);
        dim3 numberOfBlocks_connect(1, h_allEvents[i].numberOfLayerPairs);
        dim3 numberOfBlocks_find(1, h_allEvents[i].numberOfRootLayerPairs);
// KERNELS
//        debug_input_data<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex], &d_doublets[d_firstLayerPairInEvt], &d_layers[d_firstLayerInEvt],d_regionParams,  maxNumberOfHits );
        kernel_create<<<numberOfBlocks_create,256,0,streams[streamIndex]>>>(&d_events[streamIndex], &d_doublets[d_firstLayerPairInEvt],
                &d_layers[d_firstLayerInEvt], &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets],
                &device_isOuterHitOfCell[d_firstHitInEvent], &d_foundNtuplets[streamIndex],d_regionParams,  maxNumberOfDoublets, maxNumberOfHits);

////
//        kernel_debug<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex], &d_doublets[d_firstLayerPairInEvt],
//                &d_layers[d_firstLayerInEvt], &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets],
//                &device_isOuterHitOfCell[d_firstHitInEvent], &d_foundNtuplets[streamIndex],
//                d_regionParams, theThetaCut, thePhiCut,theHardPtCut,maxNumberOfDoublets, maxNumberOfHits);
        kernel_connect<<<numberOfBlocks_connect,256,0,streams[streamIndex]>>>(&d_events[streamIndex],
                &d_doublets[d_firstLayerPairInEvt], &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets],
                &device_isOuterHitOfCell[d_firstHitInEvent], d_regionParams, theThetaCut, thePhiCut,
                theHardPtCut, maxNumberOfDoublets, maxNumberOfHits);

//        kernel_debug_connect<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex], &d_doublets[d_firstLayerPairInEvt],
//                &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets], &device_isOuterHitOfCell[d_firstHitInEvent],
//                 d_regionParams, maxNumberOfDoublets, maxNumberOfHits);
//        hipMemsetAsync(&d_foundNtuplets[streamIndex], 0, sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ), streams[streamIndex]);

        kernel_find_ntuplets<<<numberOfBlocks_find,1,0,streams[streamIndex]>>>(&d_events[streamIndex],
                &d_doublets[d_firstLayerPairInEvt], &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets],
                &d_foundNtuplets[streamIndex],&d_rootLayerPairs[maxNumberOfRootLayerPairs*streamIndex], 4 , maxNumberOfDoublets);

//        kernel_debug_find_ntuplets<<<1,1,0,streams[streamIndex]>>>(&d_events[streamIndex],
//                &d_doublets[d_firstLayerPairInEvt], &device_theCells[d_firstLayerPairInEvt*maxNumberOfDoublets],
//                &d_foundNtuplets[streamIndex],&d_rootLayerPairs[maxNumberOfRootLayerPairs*streamIndex], 4 , maxNumberOfDoublets);
        hipMemcpyAsync(&h_foundNtuplets[streamIndex], &d_foundNtuplets[streamIndex], sizeof(GPUSimpleVector<maxNumberOfQuadruplets, Quadruplet> ), hipMemcpyDeviceToHost, streams[streamIndex]);
        hipMemsetAsync(&device_isOuterHitOfCell[d_firstHitInEvent], 0, maxNumberOfLayers * maxNumberOfHits* sizeof(GPUSimpleVector<maxCellsPerHit, unsigned int>), streams[streamIndex]);

        hipStreamSynchronize(streams[streamIndex]);
        std::cout << "found quadruplets " << h_foundNtuplets[streamIndex].size() << std::endl;

// COPY OF THE RESULTS
    }

    // CLEANUP

    for(int i = 0; i < numberOfCUDAStreams; ++i)
    {
        hipStreamDestroy(streams[i]);

    }

    hipFree(device_isOuterHitOfCell);
    hipFree(d_foundNtuplets);
    hipHostFree(h_foundNtuplets);
    hipHostFree(h_regionParams);
    hipFree(d_regionParams);
    hipFree(device_theCells);

    hipFree(d_events);
    hipFree(d_indices);
    hipFree(d_doublets);
    hipFree(d_layers);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(d_rootLayerPairs);

    hipHostFree(h_allEvents);
    hipHostFree(h_layers);
    hipHostFree(h_x);
    hipHostFree(h_y);
    hipHostFree(h_z);
    hipHostFree(h_rootLayerPairs);
    hipHostFree(h_indices);
    hipHostFree(h_doublets);


    return 0;
}

